#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>
#include <png.h>
#include <fstream>

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n", __FILE__, __LINE__); \
    return EXIT_FAILURE;}} while(0)

// Kernel CUDA per calcolare il frattale Burning Ship con color smoothing
__global__ void create_fractal(int xres, int yres, int iterations, float xmin, float xmax, float ymin, float ymax, float* smooth_vals) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < xres && y < yres) {
        float cx = xmin + x * (xmax - xmin) / xres; // Mappatura della coordinata x nel piano complesso
        float cy = ymin + y * (ymax - ymin) / yres; // Mappatura della coordinata y nel piano complesso
        float zx = 0.0f, zy = 0.0f;
        int n;
        for (n = 0; n < iterations; ++n) {
            float x_temp = zx * zx - zy * zy + cx;
            zy = fabs(2.0f * zx * zy) + cy;
            zx = fabs(x_temp);
            if (zx * zx + zy * zy > 4.0f)
                break;
        }
        if (n < iterations) {
            float log_zn = log(zx * zx + zy * zy) / 2.0f;
            float nu = log(log_zn / log(2.0f)) / log(2.0f);
            smooth_vals[y * xres + x] = n + 1 - nu;
        } else {
            smooth_vals[y * xres + x] = n;
        }
    }
}

// Funzione per mappare i valori smooth in colori RGB
void map_to_color(float smooth_val, int iterations, png_bytep color) {
    float t = smooth_val / iterations;
    int r = (int)(9 * (1 - t) * t * t * t * 255);
    int g = (int)(15 * (1 - t) * (1 - t) * t * t * 255);
    int b = (int)(8.5 * (1 - t) * (1 - t) * (1 - t) * t * 255);
    color[0] = (png_byte)r;
    color[1] = (png_byte)g;
    color[2] = (png_byte)b;
}

// Funzione per salvare l'immagine PNG
void save_png(const char* filename, int xres, int yres, float* smooth_vals, int iterations) {
    FILE *fp = fopen(filename, "wb");
    if (!fp) {
        std::cerr << "Errore: Impossibile aprire il file " << filename << " per la scrittura." << std::endl;
        return;
    }

    png_structp png_ptr = png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    if (!png_ptr) {
        std::cerr << "Errore: png_create_write_struct fallito." << std::endl;
        fclose(fp);
        return;
    }

    png_infop info_ptr = png_create_info_struct(png_ptr);
    if (!info_ptr) {
        std::cerr << "Errore: png_create_info_struct fallito." << std::endl;
        png_destroy_write_struct(&png_ptr, NULL);
        fclose(fp);
        return;
    }

    if (setjmp(png_jmpbuf(png_ptr))) {
        std::cerr << "Errore durante la scrittura del file PNG." << std::endl;
        png_destroy_write_struct(&png_ptr, &info_ptr);
        fclose(fp);
        return;
    }

    png_init_io(png_ptr, fp);

    // Impostazioni PNG
    png_set_IHDR(png_ptr, info_ptr, xres, yres, 8, PNG_COLOR_TYPE_RGB,
                 PNG_INTERLACE_NONE, PNG_COMPRESSION_TYPE_DEFAULT, PNG_FILTER_TYPE_DEFAULT);

    png_bytep row_pointers[yres];
    png_byte *image_data = (png_byte *)malloc(3 * xres * yres * sizeof(png_byte));
    for (int y = 0; y < yres; ++y) {
        for (int x = 0; x < xres; ++x) {
            png_bytep color = &image_data[(y * xres + x) * 3];
            map_to_color(smooth_vals[y * xres + x], iterations, color);
        }
        row_pointers[y] = &image_data[y * xres * 3];
    }

    // Scrittura dell'immagine
    png_set_rows(png_ptr, info_ptr, row_pointers);
    png_write_png(png_ptr, info_ptr, PNG_TRANSFORM_IDENTITY, NULL);

    // Pulizia
    png_destroy_write_struct(&png_ptr, &info_ptr);
    fclose(fp);
    free(image_data);

    std::cout << "Immagine salvata correttamente come " << filename << std::endl;
}

int main() {
    // Parametri
    int xres = 12800;
    int yres = 9600;
    int iterations = 100000;

    // Limiti del piano complesso per lo zoom
    float xmin = -2.5f, xmax = -2.0f;
    float ymin = -2.0f, ymax = 0.0f;

    // Allocazione memoria per i valori smooth
    size_t size = xres * yres * sizeof(float);
    float* smooth_vals;
    CUDA_CALL(hipMallocManaged(&smooth_vals, size));

    // Dimensioni dei blocchi e delle griglie CUDA
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((xres + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (yres + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Esecuzione del kernel
    create_fractal<<<numBlocks, threadsPerBlock>>>(xres, yres, iterations, xmin, xmax, ymin, ymax, smooth_vals);
    CUDA_CALL(hipGetLastError());
    CUDA_CALL(hipDeviceSynchronize());

    // Salvataggio dell'immagine PNG
    save_png("burning_ship.png", xres, yres, smooth_vals, iterations);

    // Pulizia della memoria
    CUDA_CALL(hipFree(smooth_vals));

    return 0;
}

